#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N = 100

__global__ void dotproduct( float *a, float *b, float *c ) {
	const int threadsPerBlock = 100;
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float acc = 0;
    while (tid < N) {
        acc += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    } 
    cache[cacheIndex] = acc;  
    __syncthreads(); // assure, that all threads in the block did their write

    int i = blockDim.x/2; // reduction: threadsPerBlock must be a power of 2
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main(void) {
	float *a1_h, *a2_h, *a3_h, *a1_d, *a2_d, *a3_d;
	const int N = 100;
	size_t size = N * sizeof(float);
	
	a1_h = (float *)malloc(size);
	a2_h = (float *)malloc(size);								
	a3_h = (float *)malloc(size);
	for (int i=0; i<N; i++) {
		a1_h[i] = (float)i;
		a2_h[i] = (float)i+5;
		a3_h[i] = (float)i*0.5;
	}
	
//	hipEvent_t start, stop;
//	HANDLE_ERROR( hipEventCreate( &start ) ); 
//	HANDLE_ERROR( hipEventCreate( &stop ) ); 
//	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	hipMalloc((void **)) &a1_d, size);	
	hipMalloc((void **)) &a2_d, size);
	hipMalloc((void **)) &a3_d, size);
	hipMemcpy(a1_d, a1_h, size, hipMemcpyHostToDevice);		
	hipMemcpy(a2_d, a2_h, size, hipMemcpyHostToDevice);		
	hipMemcpy(a3_d, a3_h, size, hipMemcpyHostToDevice);		
	
	int block_size = 4;
	int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	dotproduct <<< n_blocks, block_size >>> (a1_d, a2_d, a3_d);
	free(a1_h, a2_h, a3_h); hipFree(a1_d, a2_d, a3_d);

//	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
//	HANDLE_ERROR( hipEventSynchronize( stop ) );
//	float   elapsedTime;
//	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
//	printf( "Time for ...:  %3.1f ms\n", elapsedTime );
	
//	HANDLE_ERROR( hipEventDestroy( start ) );
//	HANDLE_ERROR( hipEventDestroy( stop ) );
}
